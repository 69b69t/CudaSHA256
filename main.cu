#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include "sha256.cuh"


__global__ void sha256_cuda(JOB ** jobs, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// perform sha256 calculation here
	if (i < n){
		SHA256_CTX ctx;
		sha256_init(&ctx);
		sha256_update(&ctx, jobs[i]->data, jobs[i]->size);
		sha256_final(&ctx, jobs[i]->digest);
	}
}

void pre_sha256() {
	// compy symbols
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));
}


void runJobs(JOB ** jobs, int n)
{
	int blockSize = 4;
	int numBlocks = (n + blockSize - 1) / blockSize;
	sha256_cuda <<< numBlocks, blockSize >>> (jobs, n);
}

void print_jobs(JOB ** jobs, int n) {
	printf("@ %p jobs  \n", jobs);
	for (int i = 0; i < n; i++)
	{
		printf("@ %p JOB[%i] \n", jobs[i], i);
		printf("\t @ 0x%p data = %x \n", jobs[i]->data, (jobs[i]->data == 0)? 0 : jobs[i]->data[0]);
		printf("\t @ 0x%p size = %llu \n", &(jobs[i]->size), jobs[i]->size);
		printf("\t @ 0x%p digest = %s \n------\n", jobs[i]->digest, print_sha(jobs[i]->digest));
	}
}

JOB * JOB_init(BYTE * data, long size) {
	JOB * j;
	checkCudaErrors(hipMallocManaged(&j, sizeof(JOB)));	//j = (JOB *)malloc(sizeof(JOB));
	checkCudaErrors(hipMallocManaged(&(j->data), size));
	j->data = data;
	j->size = size;
	for (int i = 0; i < 64; i++)
	{
		j->digest[i] = 0xff;
	}
	return j;
}

void print_usage(){
	printf("/.CuadaSHA256 <file>");
}


BYTE * get_file_data(char * fname, unsigned long * size){
	FILE * f = 0;
	BYTE * buffer = 0;
	unsigned long fsize = 0;

	f = fopen(fname, "rb");
	if (!f){
		fprintf(stderr, "Unable to open %s\n", fname);
		return 0;
	}
	fflush(f);

	if (fseek(f, 0, SEEK_END)){
		fprintf(stderr, "Unable to fseek %s\n", fname);
		return 0;
	}
	fflush(f);
	fsize = ftell(f);
	rewind(f);

	//buffer = (char *)malloc((fsize+1)*sizeof(char));
	checkCudaErrors(hipMallocManaged(&buffer, (fsize+1)*sizeof(char)));
	fread(buffer, fsize, 1, f);
	fclose(f);
	*size = fsize;
	printf("fsize: %lu\n", fsize);
	return buffer;
}

int main(int argc, char **argv)
{
	int i;
	unsigned long temp;
	char * a_file;
	BYTE * buff;
	char option, index;

	// parse input
    while ((option = getopt(argc, argv,"f:")) != -1)
        switch (option) {
             case 'f' :
            	 a_file = optarg;
                 break;
             default:
            	 break;
        }

    // get number of arguments = files = jobs
    int n = argc - optind;

    JOB ** jobs;
    checkCudaErrors(hipMallocManaged(&jobs, n * sizeof(JOB *)));

    // iterate over file list
    for (i = 0, index = optind; index < argc; index++, i++){
        printf ("Non-option argument '%s'\n", argv[index]);
        buff = get_file_data(argv[index], &temp);
        jobs[i] = JOB_init(buff, temp);
    }

    print_jobs(jobs, n);
	pre_sha256();
	runJobs(jobs, n);
	hipDeviceSynchronize();
	print_jobs(jobs, n);

	hipDeviceReset();
    return 0;
}
